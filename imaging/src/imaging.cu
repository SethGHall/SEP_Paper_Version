
// Copyright 2019 Adam Campbell, Seth Hall, Andrew Ensor
// Copyright 2019 High Performance Computing Research Laboratory, Auckland University of Technology (AUT)

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:

// 1. Redistributions of source code must retain the above copyright notice,
// this list of conditions and the following disclaimer.

// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.

// 3. Neither the name of the copyright holder nor the names of its
// contributors may be used to endorse or promote products derived from this
// software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.

#include "../imaging.h"
#include "../controller.h"
#include "../restorer.h"

/*
	Temporarily executing msmfs solver - This is just a placeholder and cannot major cycle nor clean at this stage
*/
int execute_imaging_pipeline_msmfs(Config *config, Host_Mem_Handles *host_mem)
{
	hipDeviceReset();
	
	Device_Mem_Handles device_mem;
	init_gpu_mem(&device_mem);
	
	Timing timers;
	init_timers(&timers);
	
	//Generate  PSFs for MSMFS cleaning - added later 
	
#if SOLVER == NIFTY_GRIDDING
	int  taylor = (2 * config->mf_num_moments) - 1;
	config->enable_psf = true;
	msmfs_nifty_gridding_execute(config, host_mem, &device_mem, &timers);
	host_mem->h_psf = (PRECISION*) calloc(taylor*config->psf_size*config->psf_size, sizeof(PRECISION));
	pull_msmfs_image_cube(config, host_mem, &device_mem);
	save_taylor_term_planes(config, host_mem->h_psf, taylor, config->psf_size);
	msmfs_nifty_clean_up(&device_mem);
	
	config->enable_psf = false;
	taylor = config->mf_num_moments;
	gains_apply_execute(config, host_mem, &device_mem, &timers);
	msmfs_nifty_gridding_execute(config, host_mem, &device_mem, &timers);
	host_mem->dirty_image = (PRECISION*) calloc(taylor*config->image_size*config->image_size, sizeof(PRECISION));
	pull_msmfs_image_cube(config, host_mem, &device_mem);
	save_taylor_term_planes(config, host_mem->dirty_image, taylor, config->image_size);
	msmfs_nifty_clean_up(&device_mem);
#else 
	int  taylor = (2 * config->mf_num_moments) - 1;
	config->enable_psf = true;
	gridding_execute_msmfs(config,host_mem,&device_mem, &timers);
	host_mem->h_psf = (PRECISION*) calloc(taylor*config->psf_size*config->psf_size, sizeof(PRECISION));
	pull_msmfs_image_cube(config, host_mem, &device_mem);
	save_taylor_term_planes(config, host_mem->h_psf, taylor, config->psf_size);
	msmfs_gridding_clean_up(&device_mem);
	config->enable_psf = false;
	
	//Generate  Dirty Images for MSMFS cleaning - added later 
	taylor = config->mf_num_moments;
	gains_apply_execute(config, host_mem, &device_mem, &timers);
	gridding_execute_msmfs(config,host_mem,&device_mem, &timers);
	host_mem->dirty_image = (PRECISION*) calloc(taylor*config->image_size*config->image_size, sizeof(PRECISION));
	pull_msmfs_image_cube(config, host_mem, &device_mem);
	save_taylor_term_planes(config, host_mem->dirty_image, taylor, config->image_size);
	msmfs_gridding_clean_up(&device_mem);
#endif	
	
	
	clean_up_device(&device_mem); 
	
	return EXIT_SUCCESS;
}


void save_taylor_term_planes(Config *config, PRECISION *host_image, int taylor_terms, int dim)
{
	
	char buffer[MAX_LEN_CHAR_BUFF * 4];
	
	for(int t=0;t<taylor_terms;t++)
	{
		snprintf(buffer, MAX_LEN_CHAR_BUFF*4, "%s%s_Plane_%d_psf.bin", config->data_output_path,(config->enable_psf ? "psf" : "dirtyimage"), t);
		printf("UPDATE >>> Attempting to save image to %s... \n\n", buffer);
		FILE *f = fopen(buffer, "wb");
	
		int saved = fwrite(host_image+t*dim*dim, sizeof(PRECISION), dim * dim, f);
		printf(">>> GRID DIMS IS : %d\n", dim);
		printf(">>> SAVED TO FILE: %d\n", saved);
		fclose(f);
	}
}


int execute_imaging_pipeline(Config *config, Host_Mem_Handles *host_mem)
{	
	// Destroy any hanging device mem allocations
	hipDeviceReset();

	Timing timers;
	init_timers(&timers);

	Device_Mem_Handles device_mem;
	init_gpu_mem(&device_mem);

	// IMPLEMENTING: Perform visibility weighting here (required for PSF and gridding)
	
	//ToDo - taken out weight  scaling until we develop a better MSMFS solution
	
	visibility_weighting_execute(config, host_mem, &device_mem); 

	// Create PSF HERE will depend what solver we are using
	generate_psf(config, host_mem, &device_mem, &timers);
	//extract_pipeline_image(host_mem->h_psf, device_mem.d_psf, config->image_size);
	 
	 //Scale pixels by sum of visibility weights used during gridding
	//normalize_image_for_weighting(config->image_size, host_mem->dirty_image, config->visibility_scaled_weights_sum);
	 //save_image_to_file(config, host_mem->h_psf, "residual_image.bin", 0);
//exit(0);
	for(int cycle = 0; cycle < config->num_major_cycles; cycle++)
	{
		if(config->perform_gain_calibration && cycle == config->number_cal_major_cycles)
		{	
			config->perform_gain_calibration = false;
			config->num_sources = 0;
			//reset host and device mem predicted
			if(host_mem->visibilities != NULL)
				memset(host_mem->visibilities,0,config->num_host_visibilities * sizeof(VIS_PRECISION2));
		}
		if(cycle < config->number_cal_major_cycles)
			printf("UPDATE >>> Executing Calibration: Major cycle number %d...\n\n", cycle);
		else
			printf("UPDATE >>> Executing Imaging: Major cycle number %d...\n\n", cycle);
		// Gains Application / Subtraction
		gains_apply_execute(config, host_mem, &device_mem, &timers);

		// Gridding / FFT / Convolution Correction

#if SOLVER == NIFTY_GRIDDING		
		printf("UPDATE >>> EXECUTING NIFTY GRIDDER...\n\n");
		nifty_gridding_execute(config, host_mem, &device_mem, &timers);
#else 
		printf("UPDATE >>> EXECUTING W_PROJECTION GRIDDER...\n\n");
		gridding_execute(config, host_mem, &device_mem, &timers);
#endif
		if(config->save_dirty_image)
		{	
			extract_pipeline_image(host_mem->dirty_image, device_mem.d_image, config->image_size);
			// Scale pixels by sum of visibility weights used during gridding
			// normalize_image_for_weighting(config->image_size, host_mem->dirty_image, config->visibility_scaled_weights_sum);
			save_image_to_file(config, host_mem->dirty_image, "dirty_image.bin", cycle);
		}

		// image restoration
		if (cycle == config->num_major_cycles-1)
		{
			printf("AG UPDATE >>> Performing image restoration...\n\n");
			//do_image_restoration(config, host_mem, &device_mem);
		}
		else
		{
			// Deconvolution
			deconvolution_execute(config, host_mem, &device_mem, &timers);

			printf("Number of sources found is %d \n\n",config->num_sources);
			
			if(config->save_residual_image)
			{	
				extract_pipeline_image(host_mem->residual_image, device_mem.d_image, config->image_size);
				save_image_to_file(config, host_mem->residual_image, "residual_image.bin", cycle);
			}

			if(config->save_extracted_sources && config->num_sources > 0)
			{
				extract_extracted_sources(config, host_mem, &device_mem);
				save_extracted_sources(host_mem->h_sources, config->num_sources, config->data_output_path,config->imaging_output_id,
					"model_sources.bin", cycle);
			}

			#if PREDICT == DFT_PREDICTION
				// DFT
				printf("UPDATE >>> PREDICT PIPELINE - DFT  ...\n");
				dft_execute(config, host_mem, &device_mem, &timers);
			#elif PREDICT == NIFTY_GRIDDING
				printf("UPDATE >>> PREDICT PIPELINE - NIFTY_DEGRIDDING  ...\n");
				//plot model sources on a grid.
				execute_source_list_to_image(config,host_mem,&device_mem);
				
				nifty_degridding_execute(config, host_mem, &device_mem, &timers);
			#else
				printf("UPDATE >>> PREDICT PIPELINE - W_PROJECTION DEGRIDDING...\n");
				//execute_source_list_to_image(config,host_mem,&device_mem);
				degridding_execute(config,host_mem,&device_mem,&timers);
			#endif

			// Gain Calibration
			if(config->perform_gain_calibration)
			{	
				gain_calibration_execute(config, host_mem, &device_mem, &timers);

				if(config->save_estimated_gains)
				{	extract_pipeline_gains(config, host_mem, &device_mem);
					rotateAndOutputGains(config, host_mem, cycle);
				}
			}
		}
	}

	report_timings(&timers);
	clean_up_device(&device_mem);
	return EXIT_SUCCESS;
}

void generate_psf(Config *config, Host_Mem_Handles *host, Device_Mem_Handles *device, Timing *timers)
{
	config->enable_psf = true;
	#if SOLVER == NIFTY_GRIDDING
		nifty_psf_execute(config, host, device, timers);
	#else //SOLVER MUST BE W_PROJ (FOR NOW)
		psf_execute(config, host, device);
	#endif
	config->enable_psf = false;
}

void normalize_image_for_weighting(const int grid_size, PRECISION *image, PRECISION weighted_sum)
{	
	// Avoid divide by zero
	if(weighted_sum > 0.0)
		for(int i = 0; i < grid_size * grid_size; i++)
			image[i] /= weighted_sum;
}

void save_psf_to_file(Config *config, PRECISION *image, const char *file_name, int start_x, int start_y, int range_x, int range_y)
{
	char buffer[MAX_LEN_CHAR_BUFF * 2];
	snprintf(buffer, MAX_LEN_CHAR_BUFF, "%s%s", config->data_output_path, file_name);
	printf("UPDATE >>> Attempting to save PSF to %s... \n\n", buffer);

	FILE *f = fopen(buffer, "w");

	if(f == NULL)
	{	
		printf(">>> ERROR: Unable to save image to file %s, check file/folder structure exists...\n\n", buffer);
		return;
	}

    for(int row = start_y; row < start_y + range_y; ++row)
    {
    	for(int col = start_x; col < start_x + range_x; ++col)
        {
            PRECISION pixel = image[row * config->grid_size + col];

			#if SINGLE_PRECISION
            	fprintf(f, "%f ", pixel);
			#else
            	fprintf(f, "%lf ", pixel);
			#endif  
        }
        fprintf(f, "\n");
    }
    fclose(f);
}

void init_gpu_mem(Device_Mem_Handles *device)
{
    (*device).d_gains           = NULL;
    (*device).d_kernels         = NULL;
    (*device).d_kernel_supports = NULL;
    (*device).d_image           = NULL;
	(*device).d_uv_grid         = NULL;
	(*device).d_weight_map      = NULL;
    (*device).d_vis_uvw_coords  = NULL;
    (*device).d_vis_weights     = NULL;
    (*device).d_visibilities    = NULL;
    (*device).d_prolate         = NULL;
    (*device).d_sources         = NULL;
    (*device).d_psf             = NULL;
    (*device).d_max_locals      = NULL;
    (*device).d_measured_vis    = NULL;
    (*device).d_receiver_pairs  = NULL;
    (*device).fft_plan          = NULL;
    (*device).d_w_grid_stack	= NULL;
}


void clean_up_device(Device_Mem_Handles *device)
{
	printf("UPDATE >>> Cleaning up all allocated device memory...\n");
    if((*device).d_gains)           CUDA_CHECK_RETURN(hipFree((*device).d_gains));
    if((*device).d_kernels)         CUDA_CHECK_RETURN(hipFree((*device).d_kernels));
    if((*device).d_kernel_supports) CUDA_CHECK_RETURN(hipFree((*device).d_kernel_supports));
    if((*device).d_image)           CUDA_CHECK_RETURN(hipFree((*device).d_image));
	if((*device).d_uv_grid)         CUDA_CHECK_RETURN(hipFree((*device).d_uv_grid));
	if((*device).d_vis_weights)     CUDA_CHECK_RETURN(hipFree((*device).d_vis_weights));
    if((*device).d_vis_uvw_coords)  CUDA_CHECK_RETURN(hipFree((*device).d_vis_uvw_coords));
    if((*device).d_weight_map)      CUDA_CHECK_RETURN(hipFree((*device).d_weight_map));
    if((*device).d_visibilities)    CUDA_CHECK_RETURN(hipFree((*device).d_visibilities));
    if((*device).d_prolate)         CUDA_CHECK_RETURN(hipFree((*device).d_prolate));
    if((*device).d_sources)         CUDA_CHECK_RETURN(hipFree((*device).d_sources));
    if((*device).d_psf)             CUDA_CHECK_RETURN(hipFree((*device).d_psf));
    if((*device).d_max_locals)      CUDA_CHECK_RETURN(hipFree((*device).d_max_locals));
    if((*device).d_measured_vis)    CUDA_CHECK_RETURN(hipFree((*device).d_measured_vis));
    if((*device).d_receiver_pairs)  CUDA_CHECK_RETURN(hipFree((*device).d_receiver_pairs));
    if((*device).d_w_grid_stack)  	CUDA_CHECK_RETURN(hipFree((*device).d_w_grid_stack));
    if((*device).fft_plan)          free((*device).fft_plan);

    (*device).d_gains           = NULL;
    (*device).d_kernels         = NULL;
    (*device).d_kernel_supports = NULL;
    (*device).d_image           = NULL;
	(*device).d_uv_grid         = NULL;
	(*device).d_weight_map      = NULL;
    (*device).d_vis_uvw_coords  = NULL;
    (*device).d_vis_weights     = NULL;
    (*device).d_visibilities    = NULL;
    (*device).d_prolate         = NULL;
    (*device).d_sources         = NULL;
    (*device).d_psf             = NULL;
    (*device).d_max_locals      = NULL;
    (*device).d_measured_vis    = NULL;
    (*device).d_receiver_pairs  = NULL;
    (*device).fft_plan          = NULL;
    (*device).d_w_grid_stack	= NULL;

}

void init_timers(Timing *timers)
{
	init_timer(&(timers->gridder));
	init_timer(&(timers->nifty_solve_stack));
	init_timer(&(timers->ifft));
	init_timer(&(timers->solve_correction));

	init_timer(&(timers->degridder));
	init_timer(&(timers->nifty_predict_stack));
	init_timer(&(timers->fft));
	init_timer(&(timers->predict_correction));

	init_timer(&(timers->deconvolution));

	init_timer(&(timers->dft));

	init_timer(&(timers->gain_subtraction));
	init_timer(&(timers->gain_calibration));

	init_timer(&(timers->solver));
	init_timer(&(timers->solver_data_ingress));

	init_timer(&(timers->predict));
	init_timer(&(timers->predict_data_ingress));
	init_timer(&(timers->predict_data_egress));
}

void report_timings(Timing *timers)
{
	printf(">>> BULLSIT...\n");
	print_timer(&(timers->gain_subtraction), "Gain Subtraction");
	print_timer(&(timers->gain_calibration), "Gain Calibration");
	print_timer(&(timers->deconvolution), "Deconvolution");
	print_timer(&(timers->dft), "Direct Fourier Transform");

	printf(">>> SOLVER TIMING...\n");
	print_timer(&(timers->solver), "Solver");
	print_timer(&(timers->solver_data_ingress), "Solver Data Ingress");
	print_timer(&(timers->gridder), "Gridding");
	print_timer(&(timers->nifty_solve_stack), "NIFTy Solver Stacking");
	print_timer(&(timers->ifft), "FFT (inverse)");
	print_timer(&(timers->solve_correction), "Solver Convolution Correction");

	printf(">>> PREDICT TIMING...\n");
	print_timer(&(timers->predict), "Predict");
	print_timer(&(timers->predict_data_ingress), "Predict Data Ingress");
	print_timer(&(timers->degridder), "Degridding");
	print_timer(&(timers->nifty_predict_stack), "NIFTy Predict Stacking");
	print_timer(&(timers->fft), "FFT");
	print_timer(&(timers->predict_correction), "Predict Convolution Correction");
	print_timer(&(timers->predict_data_egress), "Predict Data Egress");

	#if SOLVER == NIFTY_GRIDDING
		printf("UPDATE: Where solver=nifty+fft+cc \n");
	#else
		printf("UPDATE: Where solver=wprojection+fft+cc \n");
	#endif
	
	#if PREDICT == NIFTY_GRIDDING
		printf("UPDATE: Where predict=cc+fft+nifty \n");
	#elif PREDICT == W_PROJECTION_GRIDDING
		printf("UPDATE: Where predict=cc+fft+wprojection \n");
	#else
		printf("UPDATE: Where predict=dft \n");
	#endif
}

void extract_pipeline_image(PRECISION *host_image, PRECISION *device_image, const int grid_size)
{
	if(device_image != NULL)
	{	printf("UPDATE >>>>>>> ALLOCATING DEVICE IMAGE FOR EXTRACTION \n\n");
		int grid_square = grid_size * grid_size;
  		CUDA_CHECK_RETURN(hipMemcpy(host_image, device_image, grid_square * sizeof(PRECISION),hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
	}
}

void extract_predicted_visibilities(Config *config, Host_Mem_Handles *host, Device_Mem_Handles *device)
{
	if(device->d_visibilities != NULL)
	{
	    CUDA_CHECK_RETURN(hipMemcpy(host->visibilities, device->d_visibilities, 
        	config->num_host_visibilities * sizeof(VIS_PRECISION2), hipMemcpyDeviceToHost));
			
		
		
		
    	hipDeviceSynchronize();
	}
}

void extract_extracted_sources(Config *config, Host_Mem_Handles *host, Device_Mem_Handles *device)
{
	if(device->d_sources != NULL)
	{
		if(host->h_sources)
		{
			free(host->h_sources);
			host->h_sources = NULL;
		}

		host->h_sources = (Source*) calloc(config->num_sources, sizeof(Source));

	    CUDA_CHECK_RETURN(hipMemcpy(host->h_sources, device->d_sources, config->num_sources * sizeof(Source),
	        hipMemcpyDeviceToHost));
	    hipDeviceSynchronize();
	}
}

void extract_pipeline_gains(Config *config, Host_Mem_Handles *host, Device_Mem_Handles *device)
{
	if(device->d_gains != NULL)
	{
	    CUDA_CHECK_RETURN(hipMemcpy(host->h_gains, device->d_gains, 
        	config->num_recievers * sizeof(PRECISION2), hipMemcpyDeviceToHost));
    	hipDeviceSynchronize();
	}
}
